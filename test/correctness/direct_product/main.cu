#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <type_traits>
#include <wmma_extension.hpp>

#ifndef TEST_ARCH
#define TEST_ARCH (-1)
#endif

// #define TEST_TF32

#ifndef TEST_TF32
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 16;
using ab_type = half;
#else
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 8;
using ab_type = nvcuda::wmma::precision::tf32;
#endif

#define SMALLER_WORKING_MEMORY

using storage_t = typename mtk::wmma::detail::common::storage_t<ab_type>::type;

template <class T, class S>
__device__ __host__ T convert(const S);
template <> __device__ __host__ float convert<float, float>(const float a) {return a;}
template <> __device__ __host__ float convert<float, half >(const half  a) {return __half2float(a);}
template <> __device__ __host__ half  convert<half , float>(const float a) {return __float2half(a);}
template <> __device__ __host__ half  convert<half , half >(const half  a) {return a;}

template <unsigned CORRECTION_TERMS>
__global__ void direct_product_kernel(float* const h, const float* const u) {
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a   , M, N, K, ab_type, nvcuda::wmma::col_major> frag_a;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b   , M, N, K, ab_type, nvcuda::wmma::row_major> frag_b;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, M, N, K, float> frag_c;

	__shared__ storage_t su[N];
	__shared__ storage_t sdu[N];

	if (threadIdx.x < N) {
		const auto fv = u[threadIdx.x];
		const auto hv = mtk::wmma::detail::common::cast<ab_type>(fv);
		su[threadIdx.x] = hv;
		sdu[threadIdx.x] = convert<storage_t>(fv - convert<float>(hv));
	}

	if (CORRECTION_TERMS == 3) {
#ifdef SMALLER_WORKING_MEMORY
		mtk::wmma::make_direct_product_fragment_c3(
				frag_a,
				u
				);
		mtk::wmma::make_direct_product_fragment_c3(
				frag_b,
				u
				);
#else
		mtk::wmma::make_direct_product_fragment_c3(
				frag_a,
				su, sdu
				);
		mtk::wmma::make_direct_product_fragment_c3(
				frag_b,
				su, sdu
				);
#endif
	} else {
#ifdef SMALLER_WORKING_MEMORY
		mtk::wmma::make_direct_product_fragment(
				frag_a,
				u
				);
		mtk::wmma::make_direct_product_fragment(
				frag_b,
				u
				);
#else
		mtk::wmma::make_direct_product_fragment(
				frag_a,
				su, sdu
				);
		mtk::wmma::make_direct_product_fragment(
				frag_b,
				su, sdu
				);
#endif
	}

	nvcuda::wmma::fill_fragment(frag_c, 0.0f);

	nvcuda::wmma::mma_sync(frag_c, frag_a, frag_b, frag_c);

	nvcuda::wmma::store_matrix_sync(h, frag_c, N, nvcuda::wmma::mem_col_major);
}

template <unsigned CORRECTION_TERMS>
void test() {
	std::printf("-- direct_product test --\n");
	std::printf("arch    : %d\n", TEST_ARCH);
	std::printf("c terms : %u\n", CORRECTION_TERMS);
	float *u;
	float *h;

	hipHostMalloc(&u, sizeof(float) * N);
	hipHostMalloc(&h, sizeof(float) * N * N);

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	for (unsigned i = 0; i < N; i++) {
		u[i] = dist(mt);
	}

	hipDeviceSynchronize();
	direct_product_kernel<CORRECTION_TERMS><<<1, 32>>>(h, u);
	hipDeviceSynchronize();

	double max_error = 0.0;
	for (unsigned i = 0; i < N; i++) {
		for (unsigned j = 0; j < N; j++) {
			const double diff = static_cast<double>(u[i]) * static_cast<double>(u[j]) - static_cast<double>(h[i * N + j]);
			max_error = std::max(max_error, std::abs(diff));
		}
	}
	std::printf("error   : %e\n", max_error);
}

int main() {
	test<2>();
	test<3>();
}
