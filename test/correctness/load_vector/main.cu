#include "hip/hip_runtime.h"
#include <iostream>
#include <type_traits>
#include <wmma_extension.hpp>

#ifndef TEST_ARCH
#define TEST_ARCH (-1)
#endif

//#define TEST_TF32

#ifndef TEST_TF32
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 16;
using ab_type = half;
#else
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 8;
using ab_type = nvcuda::wmma::precision::tf32;
#endif

using storage_t = typename mtk::wmma::detail::common::storage_t<ab_type>::type;

template <class T, class S>
__device__ __host__ T convert(const S);
template <> __device__ __host__ float convert<float, float>(const float a) {return a;}
template <> __device__ __host__ float convert<float, half >(const half  a) {return __half2float(a);}
template <> __device__ __host__ half  convert<half , float>(const float a) {return __float2half(a);}
template <> __device__ __host__ half  convert<half , half >(const half  a) {return a;}

template <class T>
__device__ T m_abs(const T a) {
	if (a >= convert<T, float>(0)) return a;
	return -1;
}

template <class Use, class layout>
__global__ void test_load_vector_kernel(
		const storage_t* const src,
		const storage_t* const cor
		) {
	nvcuda::wmma::fragment<Use, M, N, K, ab_type, layout> vec_frag;
	mtk::wmma::load_vector_sync(vec_frag, src);

	nvcuda::wmma::fragment<Use, M, N, K, ab_type, layout> cor_frag;
	mtk::wmma::load_vector_sync(cor_frag, cor);

	storage_t error = convert<storage_t, float>(0.0f);
	for (unsigned i = 0; i < vec_frag.num_elements; i++) {
		error += m_abs(vec_frag.x[i] - cor_frag.x[i]);
	}
	printf("[%2u] error = %e\n", threadIdx.x, convert<float>(error));
}

template <class Use, class layout>
void test() {
	std::size_t cor_size = 0;
	std::printf("-- load_vector test --\n");
	std::printf("arch   : %d\n", TEST_ARCH);
	if (std::is_same<layout, nvcuda::wmma::col_major>::value) {
		std::printf("layout : col_major\n");
	} else {
		std::printf("layout : row_major\n");
	}
	if (std::is_same<float, ab_type>::value)
		std::printf("type   : float\n");
	if (std::is_same<half, ab_type>::value)
		std::printf("type   : half\n");
	if (std::is_same<nvcuda::wmma::precision::tf32, ab_type>::value)
		std::printf("type   : tf32\n");

	if (std::is_same<nvcuda::wmma::matrix_a, Use>::value) {
		std::printf("use    : a\n");
		cor_size = M * K;
	}
	if (std::is_same<nvcuda::wmma::matrix_b, Use>::value) {
		std::printf("use    : b\n");
		cor_size = N * K;
	}
	std::printf("size   : %lu, %lu, %lu\n", M, N, K);

	storage_t* src_mem;
	storage_t* cor_mem;

	hipHostMalloc(&src_mem, M * sizeof(storage_t));
	hipHostMalloc(&cor_mem, cor_size * sizeof(storage_t));

	for (std::size_t i = 0; i < cor_size; i++) {
		cor_mem[i] = convert<storage_t, float>(0);
	}

	for (std::size_t i = 0; i < 16; i++) {
		src_mem[i] = convert<storage_t, float>(i);
		cor_mem[i] = convert<storage_t, float>(i);
	}

	hipDeviceSynchronize();
	test_load_vector_kernel<Use, layout><<<1, 32>>>(src_mem, cor_mem);
	hipDeviceSynchronize();
}

int main() {
	test<nvcuda::wmma::matrix_a, nvcuda::wmma::col_major>();
	test<nvcuda::wmma::matrix_a, nvcuda::wmma::row_major>();

	test<nvcuda::wmma::matrix_b, nvcuda::wmma::col_major>();
	test<nvcuda::wmma::matrix_b, nvcuda::wmma::row_major>();
}
