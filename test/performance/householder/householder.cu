#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <wmma_extension/wmma_extension.hpp>

constexpr unsigned warp_size = 32;
constexpr unsigned block_size = 256;
constexpr unsigned test_count = 256;

template <unsigned DIM>
__device__ void cp_matrix(
		half2* const smem,
		const half2* const gmem
		) {
	for (unsigned i = 0; i < DIM * DIM / 2; i += warp_size) {
		const unsigned index = i + (threadIdx.x & 0x1fu);
		smem[index] = gmem[index];
	}
}

template <unsigned DIM, class HouseholderMatGen>
__global__ void batched_householder_kernel(
		half* const ptr,
		const unsigned batch_size) {
	__shared__ half smem_mat[DIM * DIM * block_size / warp_size];
	__shared__ half smem_vec[DIM * block_size / warp_size];

	half* const smem_mat_ptr = smem_mat + DIM * DIM * (threadIdx.x / warp_size);
	half* const smem_vec_ptr = smem_vec + DIM * (threadIdx.x / warp_size);

	const unsigned matrix_id = threadIdx.x + blockIdx.x * blockDim.x / warp_size;
	if (matrix_id >= batch_size) return;

	cp_matrix<DIM>(
			reinterpret_cast<half2*>(smem_mat_ptr),
			reinterpret_cast<half2*>(ptr + DIM * DIM * ((threadIdx.x + block_size / warp_size * blockIdx.x) / warp_size))
			);
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, DIM, DIM, DIM, half, nvcuda::wmma::col_major> frag_b;
	nvcuda::wmma::load_matrix_sync(frag_b, smem_mat_ptr, DIM);
	if ((threadIdx.x & 0x1f) < DIM) {
		smem_vec_ptr[(threadIdx.x & 0x1f)] = smem_mat_ptr[(threadIdx.x & 0x1f)];
	}
	__syncwarp();

	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, DIM, DIM, DIM, half, nvcuda::wmma::col_major> frag_a;
	HouseholderMatGen{}(frag_a, smem_mat_ptr, smem_vec_ptr);

	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, DIM, DIM, DIM, half> frag_c;
	nvcuda::wmma::fill_fragment(frag_c, 0.f);

	nvcuda::wmma::mma_sync(frag_c, frag_a, frag_b, frag_c);

	nvcuda::wmma::store_matrix_sync(smem_mat_ptr, frag_c, DIM, nvcuda::wmma::mem_col_major);
	cp_matrix<DIM>(
			reinterpret_cast<half2*>(ptr + DIM * DIM * ((threadIdx.x + block_size / warp_size * blockIdx.x) / warp_size)),
			reinterpret_cast<half2*>(smem_mat_ptr)
			);
}

template <unsigned DIM>
struct HouseholderMatGenWMMA {
	__device__ void operator()(
			nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, DIM, DIM, DIM, half, nvcuda::wmma::col_major>& frag,
			half* const smem_mat,
			const half* const smem_vec
			) const {
		for (unsigned i = 0; i < DIM * DIM; i += warp_size) {
			const unsigned index = i + (threadIdx.x & 0x1fu);
			const unsigned m = index % DIM;
			const unsigned n = index / DIM;

			auto v = smem_vec[m] * smem_vec[n] * __float2half(-2.f);
			if (m == n) {
				v += __float2half(1.f);
			}
		}
		__syncwarp();
		nvcuda::wmma::load_matrix_sync(frag, smem_mat, DIM);
	}
};

template <unsigned DIM>
struct HouseholderMatGenWMMAe {
	__device__ void operator()(
			nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, DIM, DIM, DIM, half, nvcuda::wmma::col_major>& frag,
			half* const smem_mat,
			const half* const smem_vec
			) const {
		mtk::wmma::foreach_ij<decltype(frag)>(
				[&](const unsigned *list, const unsigned list_size, const unsigned i, const unsigned j) {
					auto v = smem_vec[i] * smem_vec[j] * __float2half(-2.f);
					if (j == i) {
						v += __float2half(1.f);
					}
					for (unsigned f = 0; f < list_size; f++) {
						frag.x[f] = v;
					}
				});
	}
};

template <class T>
std::string get_class_name();
template <> std::string get_class_name<HouseholderMatGenWMMA <16>>() {return "wmma_16";}
template <> std::string get_class_name<HouseholderMatGenWMMAe<16>>() {return "wmmae_16";}

template <unsigned DIM, class HouseholderMatGen>
void batched_householder(
		half* const ptr,
		const unsigned batch_size
		) {
	const unsigned grid_size = (batch_size * warp_size + block_size - 1) / block_size;
	batched_householder_kernel<DIM, HouseholderMatGen><<<grid_size, block_size>>>(ptr, batch_size);
}

template <unsigned DIM, class HouseholderMatGen>
void test_batched_kernel(
		const unsigned batch_size
		) {
	half* input_matrix;
	hipMalloc(&input_matrix, sizeof(half) * DIM * DIM * batch_size);
	const auto start_clock = std::chrono::system_clock::now();
	for (unsigned c = 0; c < test_count; c++) {
		batched_householder<DIM, HouseholderMatGen>(
				input_matrix,
				batch_size);
	}
	hipDeviceSynchronize();
	const auto end_clock = std::chrono::system_clock::now();
	hipFree(input_matrix);

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() / static_cast<double>(test_count) * 1e-6;

	std::printf("%u,%s,%e\n", batch_size, get_class_name<HouseholderMatGen>().c_str(), elapsed_time);
}

int main() {
	std::printf("batch_size,api,time\n");
	for (unsigned i = 10; i <= 24; i++) {
		test_batched_kernel<16, HouseholderMatGenWMMA <16>>(1u << i);
		test_batched_kernel<16, HouseholderMatGenWMMAe<16>>(1u << i);
	}
}
